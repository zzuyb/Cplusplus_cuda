#include<stdio.h>

int main(){
	hipDeviceProp_t prop;
	//获取参数
	hipGetDeviceProperties(&prop,0);
	printf("---GPU设备基本信息---\n");
	printf("名称：%s\n",prop.name);
	printf("时钟频率：%d\n",prop.clockRate);
	printf("版本号：%d.%d\n",prop.major,prop.minor);
	printf("总显存大小：%ld\n",prop.totalGlobalMem);
	printf("常量内存大小：%ld\n",prop.totalConstMem);
	printf("微处理器数量：%d\n",prop.multiProcessorCount);
	printf("每个线程块的共享内存大小：%ld\n",prop.sharedMemPerBlock);
	printf("每个线程块中的寄存器数量：%d\n",prop.regsPerBlock);
	printf("线程块最大包含的线程数：%d\n",prop.maxThreadsPerBlock);
	printf("线程块的最大维度：(%d %d %d)\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
	printf("网格的最大维度：(%d %d %d)\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
	printf("纹理内存最大维度：(%d %d %d)\n",prop.maxTexture3D[0],prop.maxTexture3D[1],prop.maxTexture3D[2]);
}